#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hipcub/hipcub.hpp>
#define N 1024


__global__ void first(double* u, double* up, int n) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i > 0 && i < n-1) && (j > 0 && j < n-1))
        up[i*n + j] = 0.25 * (u[i*n + j - 1] + u[i*n + j + 1] + u[(i - 1)*n + j] + u[(i + 1)*n + j]);
}


__global__ void second(double* u, double* up, double* arr, int n) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= 0 && i < n) && (j >= 0 && j < n))
        arr[i*n + j] = up[i*n + j] - u[i*n + j];
}


int main() {

    double* u = (double*)calloc(N*N, sizeof(double));
    double* up = (double*)calloc(N*N, sizeof(double));
    
    double x1 = 10.0;
    double x2 = 20.0;
    double y1 = 20.0;
    double y2 = 30.0;
    u[0] = up[0] = x1;
    u[N] = up[N] = x2;
    u[N * (N - 1) + 1] = up[N * (N - 1) + 1] = y1;
    u[N * N] = up[N * N] = y2;

    double step1 = 10.0/(N-1);

    for (int i = 1; i < N-1; i++) {
        u[i*N] = up[i*N] = x1 + i * step1;
        u[i] = up[i] = x1 + i * step1;
        u[(N - 1) * N + i] = up[(N - 1) * N + i] = y1 + i * step1;
        u[i * N + (N - 1)] = up[i * N + (N - 1)] = x2 + i * step1;
    }

    double* ux;
    double* upx;
    double* arrx;
        
    hipMalloc(&ux, sizeof(double)*N*N);
    hipMalloc(&upx, sizeof(double)*N*N);
    hipMalloc(&arrx, sizeof(double)*N*N);
    
    hipMemcpy(ux, u, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(upx, up, N*N*sizeof(double), hipMemcpyHostToDevice);

    dim3 BLOCK_SIZE = dim3(32, 32);
    dim3 GRID_SIZE = dim3(ceil(N/32.), ceil(N/32.));

    int itter = 0;
    double* error = (double*)calloc(1,sizeof(double));
    *error = 1;
    double* errx;
    hipMalloc(&errx, sizeof(double));
    void* store = NULL;
    size_t bytes = 0;

    hipcub::DeviceReduce::Max(store, bytes, arrx, errx, N*N);
    hipMalloc(&store, bytes);

    itter = 0;
    int graphitt = 1000000/100;

    hipStream_t stream;
    hipStreamCreate(&stream);

    bool flag = false;
    hipGraph_t graph;
    hipGraphExec_t instance;

    while(*error > 1e-6 && itter < graphitt)
    {	
	itter += 2;

        if(!flag)
        {
            hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
            for(int i = 0; i < 100; i ++)
            {
                first<<<GRID_SIZE,BLOCK_SIZE,0,stream>>>(upx, ux, N);
                first<<<GRID_SIZE, BLOCK_SIZE,0,stream>>>(ux, upx, N);
            }
            hipStreamEndCapture(stream, &graph);
            hipGraphInstantiate(&instance, graph, NULL, NULL, 0);

            flag=true;
        }
        hipGraphLaunch(instance, stream);
        hipStreamSynchronize(stream);

        printf("%d %e\n", itter*100, *error);
        *error = 0;
        double* buf = ux;
        ux = upx;
        upx = buf;

        second<<<GRID_SIZE, BLOCK_SIZE, 0, stream>>>(upx, ux, arrx, N);
        hipcub::DeviceReduce::Max(store, bytes, arrx, errx, N*N, stream);
        hipMemcpyAsync(error, errx, sizeof(double), hipMemcpyDeviceToHost, stream);

        buf = ux;
        ux = upx;
        upx = buf;

        hipStreamSynchronize(stream);
    }

    free(u);
    free(up);
    hipFree(ux);
    hipFree(upx);
    hipFree(arrx);

    printf("%d %lf\n", itter, *error);

    return 0;

}
