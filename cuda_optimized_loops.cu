#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#include <hipcub/hipcub.hpp>
#define N 2048


__global__ void first(double* u, double* up, int n) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i > 0 && i < n-1) && (j > 0 && j < n-1))
        up[i*n + j] = 0.25 * (u[i*n + j - 1] + u[i*n + j + 1] + u[(i - 1)*n + j] + u[(i + 1)*n + j]);
}


__global__ void second(double* u, double* up, double* arr, int n) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if ((i >= 0 && i < n) && (j >= 0 && j < n))
        arr[i*n + j] = up[i*n + j] - u[i*n + j];
}


int main() {

    double* u = (double*)calloc(N*N, sizeof(double));
    double* up = (double*)calloc(N*N, sizeof(double));
    
    double x1 = 10.0;
    double x2 = 20.0;
    double y1 = 20.0;
    double y2 = 30.0;
    u[0] = up[0] = x1;
    u[N] = up[N] = x2;
    u[N * (N - 1) + 1] = up[N * (N - 1) + 1] = y1;
    u[N * N] = up[N * N] = y2;

    double step1 = 10.0/(N-1);

    for (int i = 1; i < N-1; i++) {
        u[i*N] = up[i*N] = x1 + i * step1;
        u[i] = up[i] = x1 + i * step1;
        u[(N - 1) * N + i] = up[(N - 1) * N + i] = y1 + i * step1;
        u[i * N + (N - 1)] = up[i * N + (N - 1)] = x2 + i * step1;
    }

    double* ux;
    double* upx;
    double* arrx;
        
    hipMalloc(&ux, sizeof(double)*N*N);
    hipMalloc(&upx, sizeof(double)*N*N);
    hipMalloc(&arrx, sizeof(double)*N*N);
    
    hipMemcpy(ux, u, N*N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(upx, up, N*N*sizeof(double), hipMemcpyHostToDevice);

    dim3 BLOCK_SIZE = dim3(32, 32);
    dim3 GRID_SIZE = dim3(ceil(N/32.), ceil(N/32.));

    int itter = 0;
    double error = 1.0;
    double* errx;
    hipMalloc(&errx, sizeof(double));
    void* store = NULL;
    size_t bytes = 0;

    while(itter < 1000000 && error > 1e-6) {
        itter++;
	if ((itter%150==0) || (itter==1)){
	    
	    error = 0.0;

	    first<<<GRID_SIZE,BLOCK_SIZE>>>(ux, upx, N);
            second<<<GRID_SIZE, BLOCK_SIZE>>>(ux, upx, arrx, N);

	    hipcub::DeviceReduce::Max(store, bytes, arrx, errx, N*N);
	    hipMalloc(&store, bytes);
	    hipcub::DeviceReduce::Max(store, bytes, arrx, errx, N*N);

            hipMemcpy(&error, errx, sizeof(double), hipMemcpyDeviceToHost);
	
            printf("%d %f\n", itter, error);
	}
	else {
	    first<<<GRID_SIZE,BLOCK_SIZE>>>(ux, upx, N);
	}

	first<<<GRID_SIZE,BLOCK_SIZE>>>(upx, ux, N);
        second<<<GRID_SIZE, BLOCK_SIZE>>>(upx, ux, arrx, N);
    }

    // Release the memory
    free(u);
    free(up);
    hipFree(ux);
    hipFree(upx);
    hipFree(arrx);

    printf("%d %lf\n", itter, error);

    return 0;
}
